#include <iostream>
#include <iomanip>
#include <cstring>
#include <chrono>
#include <hip/hip_runtime.h>

// Tipos
typedef unsigned char BYTE;
typedef unsigned int  WORD;

// ---------- ESTRUCTURA CONTEXTO MD5 ----------
typedef struct {
    BYTE data[64];
    WORD datalen;
    unsigned long long bitlen;
    WORD state[4];
} CUDA_MD5_CTX;

// ---------- MACROS MD5 ----------
#ifndef ROTLEFT
#define ROTLEFT(a,b) ((a << b) | (a >> (32-b)))
#endif

#define F(x,y,z) ((x & y) | (~x & z))
#define G(x,y,z) ((x & z) | (y & ~z))
#define H(x,y,z) (x ^ y ^ z)
#define I(x,y,z) (y ^ (x | ~z))

#define FF(a,b,c,d,m,s,t) { a += F(b,c,d) + m + t; a = b + ROTLEFT(a,s); }
#define GG(a,b,c,d,m,s,t) { a += G(b,c,d) + m + t; a = b + ROTLEFT(a,s); }
#define HH(a,b,c,d,m,s,t) { a += H(b,c,d) + m + t; a = b + ROTLEFT(a,s); }
#define II(a,b,c,d,m,s,t) { a += I(b,c,d) + m + t; a = b + ROTLEFT(a,s); }

// ---------- FUNCIONES MD5 (device) ----------
__device__ void cuda_md5_transform(CUDA_MD5_CTX *ctx, const BYTE data[]) {
    WORD a, b, c, d, m[16], i, j;

    for (i = 0, j = 0; i < 16; ++i, j += 4)
        m[i] = (data[j]) + (data[j + 1] << 8) + (data[j + 2] << 16) + (data[j + 3] << 24);

    a = ctx->state[0]; b = ctx->state[1]; c = ctx->state[2]; d = ctx->state[3];

    FF(a,b,c,d,m[0],7,0xd76aa478);  FF(d,a,b,c,m[1],12,0xe8c7b756); FF(c,d,a,b,m[2],17,0x242070db); FF(b,c,d,a,m[3],22,0xc1bdceee);
    FF(a,b,c,d,m[4],7,0xf57c0faf);  FF(d,a,b,c,m[5],12,0x4787c62a); FF(c,d,a,b,m[6],17,0xa8304613); FF(b,c,d,a,m[7],22,0xfd469501);
    FF(a,b,c,d,m[8],7,0x698098d8);  FF(d,a,b,c,m[9],12,0x8b44f7af); FF(c,d,a,b,m[10],17,0xffff5bb1);FF(b,c,d,a,m[11],22,0x895cd7be);
    FF(a,b,c,d,m[12],7,0x6b901122); FF(d,a,b,c,m[13],12,0xfd987193);FF(c,d,a,b,m[14],17,0xa679438e);FF(b,c,d,a,m[15],22,0x49b40821);

    GG(a,b,c,d,m[1],5,0xf61e2562);  GG(d,a,b,c,m[6],9,0xc040b340);  GG(c,d,a,b,m[11],14,0x265e5a51); GG(b,c,d,a,m[0],20,0xe9b6c7aa);
    GG(a,b,c,d,m[5],5,0xd62f105d);  GG(d,a,b,c,m[10],9,0x02441453);GG(c,d,a,b,m[15],14,0xd8a1e681);GG(b,c,d,a,m[4],20,0xe7d3fbc8);
    GG(a,b,c,d,m[9],5,0x21e1cde6);  GG(d,a,b,c,m[14],9,0xc33707d6);GG(c,d,a,b,m[3],14,0xf4d50d87); GG(b,c,d,a,m[8],20,0x455a14ed);
    GG(a,b,c,d,m[13],5,0xa9e3e905); GG(d,a,b,c,m[2],9,0xfcefa3f8);  GG(c,d,a,b,m[7],14,0x676f02d9); GG(b,c,d,a,m[12],20,0x8d2a4c8a);

    HH(a,b,c,d,m[5],4,0xfffa3942);  HH(d,a,b,c,m[8],11,0x8771f681);HH(c,d,a,b,m[11],16,0x6d9d6122);HH(b,c,d,a,m[14],23,0xfde5380c);
    HH(a,b,c,d,m[1],4,0xa4beea44);  HH(d,a,b,c,m[4],11,0x4bdecfa9);HH(c,d,a,b,m[7],16,0xf6bb4b60);HH(b,c,d,a,m[10],23,0xbebfbc70);
    HH(a,b,c,d,m[13],4,0x289b7ec6); HH(d,a,b,c,m[0],11,0xeaa127fa);HH(c,d,a,b,m[3],16,0xd4ef3085);HH(b,c,d,a,m[6],23,0x04881d05);
    HH(a,b,c,d,m[9],4,0xd9d4d039);  HH(d,a,b,c,m[12],11,0xe6db99e5);HH(c,d,a,b,m[15],16,0x1fa27cf8);HH(b,c,d,a,m[2],23,0xc4ac5665);

    II(a,b,c,d,m[0],6,0xf4292244);  II(d,a,b,c,m[7],10,0x432aff97); II(c,d,a,b,m[14],15,0xab9423a7);II(b,c,d,a,m[5],21,0xfc93a039);
    II(a,b,c,d,m[12],6,0x655b59c3); II(d,a,b,c,m[3],10,0x8f0ccc92); II(c,d,a,b,m[10],15,0xffeff47d);II(b,c,d,a,m[1],21,0x85845dd1);
    II(a,b,c,d,m[8],6,0x6fa87e4f);  II(d,a,b,c,m[15],10,0xfe2ce6e0);II(c,d,a,b,m[6],15,0xa3014314); II(b,c,d,a,m[13],21,0x4e0811a1);
    II(a,b,c,d,m[4],6,0xf7537e82);  II(d,a,b,c,m[11],10,0xbd3af235);II(c,d,a,b,m[2],15,0x2ad7d2bb); II(b,c,d,a,m[9],21,0xeb86d391);

    ctx->state[0] += a;
    ctx->state[1] += b;
    ctx->state[2] += c;
    ctx->state[3] += d;
}

__device__ void cuda_md5_init(CUDA_MD5_CTX *ctx) {
    ctx->datalen = 0;
    ctx->bitlen = 0;
    ctx->state[0] = 0x67452301;
    ctx->state[1] = 0xEFCDAB89;
    ctx->state[2] = 0x98BADCFE;
    ctx->state[3] = 0x10325476;
}

__device__ void cuda_md5_update(CUDA_MD5_CTX *ctx, const BYTE data[], size_t len) {
    size_t i;
    for (i = 0; i < len; ++i) {
        ctx->data[ctx->datalen++] = data[i];
        if (ctx->datalen == 64) {
            cuda_md5_transform(ctx, ctx->data);
            ctx->bitlen += 512;
            ctx->datalen = 0;
        }
    }
}

__device__ void cuda_md5_final(CUDA_MD5_CTX *ctx, BYTE hash[]) {
    size_t i = ctx->datalen;
    if (ctx->datalen < 56) {
        ctx->data[i++] = 0x80;
        while (i < 56) ctx->data[i++] = 0x00;
    } else {
        ctx->data[i++] = 0x80;
        while (i < 64) ctx->data[i++] = 0x00;
        cuda_md5_transform(ctx, ctx->data);
        memset(ctx->data, 0, 56);
    }

    ctx->bitlen += ctx->datalen * 8;
    for (int j = 0; j < 8; ++j)
        ctx->data[56 + j] = (ctx->bitlen >> (8 * j)) & 0xFF;

    cuda_md5_transform(ctx, ctx->data);
    for (i = 0; i < 4; ++i) {
        hash[i]      = (ctx->state[0] >> (8 * i)) & 0xFF;
        hash[i + 4]  = (ctx->state[1] >> (8 * i)) & 0xFF;
        hash[i + 8]  = (ctx->state[2] >> (8 * i)) & 0xFF;
        hash[i + 12] = (ctx->state[3] >> (8 * i)) & 0xFF;
    }
}

// ---------- FUNCIONES AUXILIARES ----------
__device__ int construir_mensaje(char* out, const char* base, int base_len, long nonce) {
    for (int i = 0; i < base_len; ++i) out[i] = base[i];
    int i = base_len, pos = 0;
    char buffer[20];
    long temp = nonce;
    do {
        buffer[pos++] = '0' + (temp % 10);
        temp /= 10;
    } while (temp > 0);
    for (int j = pos - 1; j >= 0; --j) out[i++] = buffer[j];
    return i;
}

__device__ bool starts_with(BYTE* hash, int hash_len, const char* prefix, int prefix_len) {
    for (int i = 0; i < prefix_len; ++i) {
        char c = prefix[i];
        char h = ((hash[i / 2] >> ((1 - (i % 2)) * 4)) & 0xF);
        h = (h < 10) ? ('0' + h) : ('a' + (h - 10));
        if (c != h) return false;
    }
    return true;
}

// ---------- KERNEL ----------
__global__ void cuda_md5_miner_kernel(
    const char* base, int base_len,
    const char* prefix, int prefix_len,
    long start_nonce, BYTE* resultado_hash,
    long* resultado_nonce, int* encontrado,
    long cantidad_total)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= cantidad_total) return;
    long nonce = start_nonce + idx;

    char mensaje[128];
    int mensaje_len = construir_mensaje(mensaje, base, base_len, nonce);

    CUDA_MD5_CTX ctx;
    BYTE hash[16];

    cuda_md5_init(&ctx);
    cuda_md5_update(&ctx, (BYTE*)mensaje, mensaje_len);
    cuda_md5_final(&ctx, hash);

    if (starts_with(hash, 16, prefix, prefix_len)) {
        if (atomicCAS(encontrado, 0, 1) == 0) {
            *resultado_nonce = nonce;
            for (int i = 0; i < 16; ++i) resultado_hash[i] = hash[i];
        }
    }
}

// ---------- HOST ----------
struct ResultadoMinado {
    std::string hash;
    long nonce;
};

ResultadoMinado* minarGPU(const std::string& prefijo, const std::string& cadena, long minNonce, long maxNonce) {
    long cantidad_nonce = maxNonce - minNonce + 1;

    char* d_base; char* d_prefijo; BYTE* d_hash_result;
    long* d_nonce_result; int* d_encontrado;

    hipMalloc(&d_base, cadena.size());
    hipMemcpy(d_base, cadena.c_str(), cadena.size(), hipMemcpyHostToDevice);

    hipMalloc(&d_prefijo, prefijo.size());
    hipMemcpy(d_prefijo, prefijo.c_str(), prefijo.size(), hipMemcpyHostToDevice);

    hipMalloc(&d_hash_result, 16);
    hipMalloc(&d_nonce_result, sizeof(long));
    hipMalloc(&d_encontrado, sizeof(int));
    hipMemset(d_encontrado, 0, sizeof(int));

    int threads = 256;
    int blocks = (cantidad_nonce + threads - 1) / threads;

    auto inicio = std::chrono::high_resolution_clock::now();

    cuda_md5_miner_kernel<<<blocks, threads>>>(
        d_base, cadena.size(), d_prefijo, prefijo.size(),
        minNonce, d_hash_result, d_nonce_result, d_encontrado, cantidad_nonce
    );
    hipDeviceSynchronize();

    auto fin = std::chrono::high_resolution_clock::now();
    auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(fin - inicio).count();

    int encontrado = 0;
    hipMemcpy(&encontrado, d_encontrado, sizeof(int), hipMemcpyDeviceToHost);

    if (encontrado) {
        long nonce_encontrado;
        BYTE hash_bytes[16];
        hipMemcpy(&nonce_encontrado, d_nonce_result, sizeof(long), hipMemcpyDeviceToHost);
        hipMemcpy(hash_bytes, d_hash_result, 16, hipMemcpyDeviceToHost);

        std::stringstream ss;
        for (int i = 0; i < 16; ++i)
            ss << std::hex << std::setw(2) << std::setfill('0') << (int)hash_bytes[i];

        std::cout << "Tiempo: " << ms << " ms\n";
        hipFree(d_base); hipFree(d_prefijo); hipFree(d_hash_result); hipFree(d_nonce_result); hipFree(d_encontrado);
        return new ResultadoMinado{ ss.str(), nonce_encontrado };
    }

    std::cout << "Tiempo: " << ms << " ms\nNo se encontró un hash válido\n";
    hipFree(d_base); hipFree(d_prefijo); hipFree(d_hash_result); hipFree(d_nonce_result); hipFree(d_encontrado);
    return nullptr;
}

int main() {
    std::string cadena = "bloque123|TxABC";
    std::string prefijo = "0000";
    long minNonce = 100000;
    long maxNonce = 200000;

    ResultadoMinado* resultado = minarGPU(prefijo, cadena, minNonce, maxNonce);
    if (resultado) {
        std::cout << "Nonce: " << resultado->nonce << "\nHash : " << resultado->hash << std::endl;
        delete resultado;
    }

    return 0;
}
